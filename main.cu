#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <hiprand/hiprand_kernel.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include "rt.cuh"
#include "hittable_list.cuh"
#include "sphere.cuh"
#include "camera.cuh"


// Property variables
struct Properties
{
    // Image properties
    const float aspect_ratio = 16.0 / 9.0;
    const int image_width = 1920; 
    const int image_height = (int)(image_width / aspect_ratio);
    int num_pixels = image_width * image_height;
    size_t fb_size = 3 * num_pixels * sizeof(float); // rgb * numpixels * size of float

    // Render properties
    const int samples_per_pixel = 100;
    const int max_depth = 10;

};

// Error checking
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}


// Write color to array
__device__ void write_color(unsigned char *fb, int pixel_index, color pixel_color, int samples_per_pixel) 
{
    float r = pixel_color.x();
    float g = pixel_color.y();
    float b = pixel_color.z();

    // Divide color by number of samples. Gamma correct.
    float scale = 1.0 / samples_per_pixel;
    r = sqrt(scale * r);
    g = sqrt(scale * g);
    b = sqrt(scale * b);

    fb[pixel_index + 0] = int(256 * clamp(r, 0.0, 0.999));
    fb[pixel_index + 1] = int(256 * clamp(g, 0.0, 0.999));
    fb[pixel_index + 2] = int(256 * clamp(b, 0.0, 0.999));
}

// Return color of pixel
__device__ vec3 ray_color(const ray& r, hittable **world, hiprandState *local_rand_state, Properties p) {
   ray cur_ray = r;
   float cur_attenuation = 1.0f;
   for(int i = 0; i < p.max_depth; i++) {
      hit_record rec;
      if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
         vec3 target = rec.p + rec.normal + random_in_unit_sphere(local_rand_state);
         cur_attenuation *= 0.5f;
         cur_ray = ray(rec.p, target-rec.p);
      }
      else {
           vec3 unit_direction = unit_vector(cur_ray.direction());
           float t = 0.5f*(unit_direction.y() + 1.0f);
           vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
           return cur_attenuation * c;
        }
      }
   return vec3(0.0,0.0,0.0); // exceeded recursion
}

// Initializing values like random values before main render
__global__ void render_init(int max_x, int max_y, hiprandState *rand_state)
{
    // x index and y index
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) 
        return;
    int pixel_index = j * max_x + i;

    // Retrieve a random value for each thread
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

// Main render
__global__ void render(unsigned char *fb, Properties p, hittable **world, hiprandState *rand_state, camera **camera)
{
    // initialize variables and random state
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= p.image_width) || (j >= p.image_height)) 
        return;
    int pixel_index = j * p.image_width * 3 + i * 3;
    int rand_index = j * p.image_width + i;
    hiprandState local_rand_state = rand_state[rand_index];

    // calculate pixel color
    color pixel_color;
    for (int s = 0; s < p.samples_per_pixel; s++)
    {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(p.image_width - 1);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(p.image_height - 1);
        ray r = (*camera)->get_ray(u, v);
        pixel_color += ray_color(r, world, &local_rand_state, p);
    }

    // write color
    write_color(fb, pixel_index, pixel_color, p.samples_per_pixel);
}

// Allocate world
__global__ void create_world(hittable **d_list, hittable **d_world, camera **d_camera)
{
    // Allocate new objects and world
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        *d_list = new sphere(vec3(0, 0, -1), 0.5);
        *(d_list + 1) = new sphere(vec3(0, -100.5, -1), 100);
        *d_world = new hittable_list(d_list, 2);
        *d_camera = new camera();
    }
}

// Deallocate world
__global__ void free_world(hittable **d_list, hittable **d_world, camera **d_camera) 
{
    delete *(d_list);
    delete *(d_list+1);
    delete *d_world;
    delete *d_camera;
}

int main()
{
    // ------------------- Variables --------------------
    Properties p;

    // Grid dimension
    int tx = 8;
    int ty = 8;

    // -------------------- World -----------------------
    hittable **d_list;
    checkCudaErrors(hipMalloc(&d_list, 2 * sizeof(hittable *)));
    hittable **d_world;
    checkCudaErrors(hipMalloc(&d_world, sizeof(hittable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc(&d_camera, sizeof(camera*)));
    create_world<<<1, 1>>>(d_list, d_world, d_camera);
    checkCudaErrors(hipDeviceSynchronize());

    // ---------------- memory allocation ---------------
    unsigned char *fb;
    hiprandState *d_rand_state;
    checkCudaErrors(hipMallocManaged(&fb, p.fb_size));
    checkCudaErrors(hipMallocManaged(&d_rand_state, p.num_pixels * sizeof(hiprandState)));

    // Run render kernel with given sizes.
    dim3 blocks(p.image_width / tx + 1, p.image_height / ty + 1);
    dim3 threads(tx, ty);


    render_init<<<blocks, threads>>>(p.image_width, p.image_height, d_rand_state);
    checkCudaErrors(hipDeviceSynchronize());
    

    // -------------------- RENDER ----------------------
    auto start = std::chrono::high_resolution_clock::now();
    render<<<blocks, threads>>>(fb, p, d_world, d_rand_state, d_camera);
    // calculate time taken to render
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cerr << "\nFinished in: " << duration.count() / 1000.0 << "ms" << std::endl;


    checkCudaErrors(hipDeviceSynchronize()); // wait for GPU to finish
    // write to jpg
    stbi_flip_vertically_on_write(true);
    stbi_write_jpg("renders/image.jpg", p.image_width, p.image_height, 3, fb, 100);
    
    // free memory
    free_world<<<1, 1>>>(d_list, d_world, d_camera);
    checkCudaErrors(hipFree(fb));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_camera));
}